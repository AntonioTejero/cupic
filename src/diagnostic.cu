/****************************************************************************
 *                                                                          *
 *    This file is part of CUPIC, a code that simulates the interaction     *
 *    between plasma and a langmuir probe using PIC techniques accelerated  *
 *    with the use of GPU hardware (CUDA extension of C/C++)                *
 *                                                                          *
 ****************************************************************************/

/****************************** HEADERS ******************************/

#include "diagnostic.h"

/********************* HOST FUNCTION DEFINITIONS *********************/

int number_of_particles(int *d_bm) 
{
  /*--------------------------- function variables -----------------------*/
  
  // host memory
  static const int ncy = init_ncy();      // number of cells in y dimension
  
  hipError_t cuError;
  int h_bm[2*ncy];
  int ini, fin;
  
  // device memory
  
  
  /*----------------------------- function body -------------------------*/
  
  // copy vector of bookmarks from device to host
  cuError = hipMemcpy (h_bm, d_bm, 2*ncy*sizeof(int), hipMemcpyDeviceToHost);
  cu_check(cuError, __FILE__, __LINE__);

  // evaluate number of particles
  ini = 0;
  fin = 2*ncy-1;

  while (h_bm[fin] < 0 && fin > 0) fin -= 2;
  while (h_bm[ini] < 0 && ini < fin) ini += 2;
  
  if (ini > fin) return 0;
  else return h_bm[fin]-h_bm[ini]+1;
}


/**********************************************************/

void particles_snapshot(particle *d_p, int * d_bm, string filename) 
{
  /*--------------------------- function variables -----------------------*/
  
  // host memory  
  particle *h_p;
  int N;
  FILE *file;
  hipError_t cuError;
  
  // device memory
  
  
  /*----------------------------- function body -------------------------*/
  
  // evaluate number of particles in the vector
  N = number_of_particles(d_bm);
  
  // allocate host memory for particle vector
  h_p = (particle *) malloc(N*sizeof(particle));
  
  // copy particle vector from device to host
  cuError = hipMemcpy (h_p, d_p, N*sizeof(particle), hipMemcpyDeviceToHost);
  cu_check(cuError, __FILE__, __LINE__);
  
  // save snapshot to file
  filename.insert(0, "../output/");
  filename.append(".dat");
  file = fopen(filename.c_str(), "w");
  
  for (int i = 0; i < N; i++) 
  {
    fprintf(file, " %.17e %.17e %.17e %.17e \n", h_p[i].x, h_p[i].y, h_p[i].vx, h_p[i].vy);
  }
  
  fclose(file);
  
  // free host memory
  free(h_p);
  
  return;
}

/**********************************************************/

void mesh_snapshot(double *d_m, string filename) 
{
  /*--------------------------- function variables -----------------------*/
  
  // host memory 
  static const int nnx = init_nnx();
  static const int nny = init_nny();
  double *h_m;
  FILE *file;
  hipError_t cuError;
  
  // device memory
  
  
  /*----------------------------- function body -------------------------*/
  
  // allocate host memory for mesh vector
  h_m = (double *) malloc(nnx*nny*sizeof(double));
  
  // copy particle vector from device to host
  cuError = hipMemcpy (h_m, d_m, nnx*nny*sizeof(double), hipMemcpyDeviceToHost);
  cu_check(cuError, __FILE__, __LINE__);
  
  // save snapshot to file
  filename.insert(0, "../output/");
  filename.append(".dat");
  file = fopen(filename.c_str(), "w");
  
  for (int i = 0; i < nnx; i++) 
  {
    for (int j = 0; j < nny; j++) 
    {
      fprintf(file, " %d %d %.17e \n", i, j, h_m[i+j*nnx]);
    }
    fprintf(file, "\n");
  }
  
  fclose(file);
  
  // free host memory
  free(h_m);
  
  return;
}

/**********************************************************/

void save_bm(int * d_bm, string filename)
{
  /*--------------------------- function variables -----------------------*/
  
  // host memory
  static const int ncy = init_ncy();      // number of cells in y dimension
  int h_bm[2*ncy];
  FILE *file;
  hipError_t cuError;
  
  // device memory
  
  
  /*----------------------------- function body -------------------------*/
  
  // copy vector of bookmarks from device to host
  cuError = hipMemcpy (h_bm, d_bm, 2*ncy*sizeof(int), hipMemcpyDeviceToHost);
  cu_check(cuError, __FILE__, __LINE__);

  // save bookmarks to file
  filename.insert(0, "../output/");
  filename.append(".dat");
  file = fopen(filename.c_str(), "w");

  for (int i = 0; i<2*ncy; i+=2)
  {
    fprintf(file, " %d %d %d \n", i, h_bm[i], h_bm[i+1]);
  }

  fclose(file);
  
  return;
}

/**********************************************************/

void save_bins(int *d_bm, particle *d_p, string filename)
{
  /*--------------------------- function variables -----------------------*/
  
  // host memory
  static const double ds = init_ds();      // spacial step
  particle *h_p;
  int N;
  FILE *file;
  hipError_t cuError;
  
  // device memory
  
  
  /*----------------------------- function body -------------------------*/
  
  // evaluate number of particles in the vector
  N = number_of_particles(d_bm);
  
  // allocate host memory for particle vector
  h_p = (particle *) malloc(N*sizeof(particle));
  
  // copy particle vector from device to host
  cuError = hipMemcpy (h_p, d_p, N*sizeof(particle), hipMemcpyDeviceToHost);
  cu_check(cuError, __FILE__, __LINE__);
  
  // save bins to file
  filename.insert(0, "../output/");
  filename.append(".dat");
  file = fopen(filename.c_str(), "w");
  
  for (int i = 0; i < N; i++) 
  {
    fprintf(file, " %d %d \n", i, int(h_p[i].y/ds));
  }
  
  fclose(file);

  //free host memory for particle vector
  free(h_p);
  
  return;
}

/******************** DEVICE KERNELS DEFINITIONS *********************/



/**********************************************************/
