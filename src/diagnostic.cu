/****************************************************************************
 *                                                                          *
 *    This file is part of CUPIC, a code that simulates the interaction     *
 *    between plasma and a langmuir probe using PIC techniques accelerated  *
 *    with the use of GPU hardware (CUDA extension of C/C++)                *
 *                                                                          *
 ****************************************************************************/

/****************************** HEADERS ******************************/

#include "diagnostic.h"

/********************* HOST FUNCTION DEFINITIONS *********************/

unsigned int number_of_particles(unsigned int *d_bm) 
{
  /*--------------------------- function variables -----------------------*/
  
  // host memory
  static const int ncy = init_ncy();      // number of cells in y dimension
  
  unsigned int h_bm[2*ncy];
  
  // device memory
  
  
  /*----------------------------- function body -------------------------*/
  
  // copy vector of bookmarks from device to host
  hipMemcpy (h_bm, d_bm, 2*ncy*sizeof(unsigned int), hipMemcpyDeviceToHost);
  
  return h_bm[2*ncy-1]-h_bm[0]+1;
}


/**********************************************************/

void snapshot(particle *d_p, unsigned int * d_bm, string filename) 
{
  /*--------------------------- function variables -----------------------*/
  
  // host memory  
  particle *h_p;
  int N;
  ofstream file;
  
  // device memory
  
  
  /*----------------------------- function body -------------------------*/
  
  // evaluate number of particles in the vector
  N = number_of_particles(d_bm);
  
  // allocate host memory for particle vector
  h_p = (particle *) malloc(N*sizeof(particle));
  
  // copy particle vector from device to host
  hipMemcpy (h_p, d_p, N*sizeof(particle), hipMemcpyDeviceToHost);
  
  // save snapshot to file
  filename.insert(0, "../output/");
  filename.append(".dat");
  file.open(filename.c_str());
  
  for (int i = 0; i < N; i++) 
  {
    file << i << " " << h_p[i].x << " " << h_p[i].y << " " << h_p[i].vx << " " << h_p[i].vy << endl;
  }
  
  file.close();
  
  return;
}

/**********************************************************/


/******************** DEVICE KERNELS DEFINITIONS *********************/



/**********************************************************/