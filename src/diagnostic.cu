/****************************************************************************
 *                                                                          *
 *    This file is part of CUPIC, a code that simulates the interaction     *
 *    between plasma and a langmuir probe using PIC techniques accelerated  *
 *    with the use of GPU hardware (CUDA extension of C/C++)                *
 *                                                                          *
 ****************************************************************************/

/****************************** HEADERS ******************************/

#include "diagnostic.h"

/********************* HOST FUNCTION DEFINITIONS *********************/

int number_of_particles(int *d_bm) 
{
  /*--------------------------- function variables -----------------------*/
  
  // host memory
  static const int ncy = init_ncy();      // number of cells in y dimension
  
  int h_bm[2*ncy];
  
  // device memory
  
  
  /*----------------------------- function body -------------------------*/
  
  // copy vector of bookmarks from device to host
  hipMemcpy (h_bm, d_bm, 2*ncy*sizeof(int), hipMemcpyDeviceToHost);
  
  return h_bm[2*ncy-1]-h_bm[0]+1;
}


/**********************************************************/

void snapshot(particle *d_p, int * d_bm, string filename) 
{
  /*--------------------------- function variables -----------------------*/
  
  // host memory  
  particle *h_p;
  int N;
  ofstream file;
  
  // device memory
  
  
  /*----------------------------- function body -------------------------*/
  
  // evaluate number of particles in the vector
  N = number_of_particles(d_bm);
  
  // allocate host memory for particle vector
  h_p = (particle *) malloc(N*sizeof(particle));
  
  // copy particle vector from device to host
  hipMemcpy (h_p, d_p, N*sizeof(particle), hipMemcpyDeviceToHost);
  
  // save snapshot to file
  filename.insert(0, "../output/");
  filename.append(".dat");
  file.open(filename.c_str());
  
  for (int i = 0; i < N; i++) 
  {
    file << i << " " << h_p[i].x << " " << h_p[i].y << " " << h_p[i].vx << " " << h_p[i].vy << endl;
  }
  
  file.close();
  
  return;
}

/**********************************************************/

void show_bm(int * d_bm)
{
  /*--------------------------- function variables -----------------------*/
  
  // host memory
  static const int ncy = init_ncy();      // number of cells in y dimension
  int h_bm[2*ncy];
  
  // device memory
  
  
  /*----------------------------- function body -------------------------*/
  
  // copy vector of bookmarks from device to host
  hipMemcpy (h_bm, d_bm, 2*ncy*sizeof(int), hipMemcpyDeviceToHost);

  // print bookmarks
  cout << "| ";
  for (int i = 0; i<2*ncy; i+=2)
  {
    cout << h_bm[i] << "," << h_bm[i+1] << " | ";
  }
  cout << endl;

  return;
}

/**********************************************************/

void save_bins(int *d_bm, particle *d_p, string filename)
{
  /*--------------------------- function variables -----------------------*/
  
  // host memory
  static const double ds = init_ds();      // spacial step
  particle *h_p;
  int N;
  ofstream file;
  
  // device memory
  
  
  /*----------------------------- function body -------------------------*/
  
  // evaluate number of particles in the vector
  N = number_of_particles(d_bm);
  
  // allocate host memory for particle vector
  h_p = (particle *) malloc(N*sizeof(particle));
  
  // copy particle vector from device to host
  hipMemcpy (h_p, d_p, N*sizeof(particle), hipMemcpyDeviceToHost);
  
  // save bins to file
  filename.insert(0, "../output/");
  filename.append(".dat");
  file.open(filename.c_str());
  
  for (int i = 0; i < N; i++) 
  {
    file << i << " " << int(h_p[i].y/ds) << endl;
  }
  
  file.close();
  
  return;
}

/******************** DEVICE KERNELS DEFINITIONS *********************/



/**********************************************************/
