/****************************************************************************
 *                                                                          *
 *    This file is part of CUPIC, a code that simulates the interaction     *
 *    between plasma and a langmuir probe using PIC techniques accelerated  *
 *    with the use of GPU hardware (CUDA extension of C/C++)                *
 *                                                                          *
 ****************************************************************************/

/****************************** HEADERS ******************************/

#include "diagnostic.h"

/********************* HOST FUNCTION DEFINITIONS *********************/

int number_of_particles(int *d_bm) 
{
  /*--------------------------- function variables -----------------------*/
  
  // host memory
  static const int ncy = init_ncy();      // number of cells in y dimension
  
  hipError_t cuError;
  int h_bm[2*ncy];
  int ini, fin;
  
  // device memory
  
  
  /*----------------------------- function body -------------------------*/
  
  // copy vector of bookmarks from device to host
  cuError = hipMemcpy (h_bm, d_bm, 2*ncy*sizeof(int), hipMemcpyDeviceToHost);
  cu_check(cuError, __FILE__, __LINE__);

  // evaluate number of particles
  ini = 0;
  fin = 2*ncy-1;

  while (h_bm[fin] < 0 && fin > 0) fin -= 2;
  while (h_bm[ini] < 0 && ini < fin) ini += 2;
  
  if (ini > fin) return 0;
  else return h_bm[fin]-h_bm[ini]+1;
}


/**********************************************************/

void particles_snapshot(particle *d_p, int * d_bm, string filename) 
{
  /*--------------------------- function variables -----------------------*/
  
  // host memory  
  particle *h_p;
  int N;
  FILE *pFile;
  hipError_t cuError;
  
  // device memory
  
  
  /*----------------------------- function body -------------------------*/
  
  // evaluate number of particles in the vector
  N = number_of_particles(d_bm);
  
  // allocate host memory for particle vector
  h_p = (particle *) malloc(N*sizeof(particle));
  
  // copy particle vector from device to host
  cuError = hipMemcpy (h_p, d_p, N*sizeof(particle), hipMemcpyDeviceToHost);
  cu_check(cuError, __FILE__, __LINE__);
  
  // save snapshot to file
  filename.insert(0, "../output/");
  filename.append(".dat");
  pFile = fopen(filename.c_str(), "w");
  
  for (int i = 0; i < N; i++) 
  {
    fprintf(pFile, " %.17e %.17e %.17e %.17e \n", h_p[i].x, h_p[i].y, h_p[i].vx, h_p[i].vy);
  }
  
  fclose(pFile);
  
  // free host memory
  free(h_p);
  
  return;
}

/**********************************************************/

void mesh_snapshot(double *d_m, string filename) 
{
  /*--------------------------- function variables -----------------------*/
  
  // host memory 
  static const int nnx = init_nnx();
  static const int nny = init_nny();
  double *h_m;
  FILE *pFile;
  hipError_t cuError;
  
  // device memory
  
  
  /*----------------------------- function body -------------------------*/
  
  // allocate host memory for mesh vector
  h_m = (double *) malloc(nnx*nny*sizeof(double));
  
  // copy particle vector from device to host
  cuError = hipMemcpy (h_m, d_m, nnx*nny*sizeof(double), hipMemcpyDeviceToHost);
  cu_check(cuError, __FILE__, __LINE__);
  
  // save snapshot to file
  filename.insert(0, "../output/");
  filename.append(".dat");
  pFile = fopen(filename.c_str(), "w");
  
  for (int i = 0; i < nnx; i++) 
  {
    for (int j = 0; j < nny; j++) 
    {
      fprintf(pFile, " %d %d %.17e \n", i, j, h_m[i+j*nnx]);
    }
    fprintf(pFile, "\n");
  }
  
  fclose(pFile);
  
  // free host memory
  free(h_m);
  
  return;
}

/**********************************************************/

void save_bm(int * d_bm, string filename)
{
  /*--------------------------- function variables -----------------------*/
  
  // host memory
  static const int ncy = init_ncy();      // number of cells in y dimension
  int h_bm[2*ncy];
  FILE *pFile;
  hipError_t cuError;
  
  // device memory
  
  
  /*----------------------------- function body -------------------------*/
  
  // copy vector of bookmarks from device to host
  cuError = hipMemcpy (h_bm, d_bm, 2*ncy*sizeof(int), hipMemcpyDeviceToHost);
  cu_check(cuError, __FILE__, __LINE__);

  // save bookmarks to file
  filename.insert(0, "../output/");
  filename.append(".dat");
  pFile = fopen(filename.c_str(), "w");

  for (int i = 0; i<2*ncy; i+=2)
  {
    fprintf(pFile, " %d %d %d \n", i, h_bm[i], h_bm[i+1]);
  }

  fclose(pFile);
  
  return;
}

/**********************************************************/

void save_bins(int *d_bm, particle *d_p, string filename)
{
  /*--------------------------- function variables -----------------------*/
  
  // host memory
  static const double ds = init_ds();      // spacial step
  particle *h_p;
  int N;
  FILE *pFile;
  hipError_t cuError;
  
  // device memory
  
  
  /*----------------------------- function body -------------------------*/
  
  // evaluate number of particles in the vector
  N = number_of_particles(d_bm);
  
  // allocate host memory for particle vector
  h_p = (particle *) malloc(N*sizeof(particle));
  
  // copy particle vector from device to host
  cuError = hipMemcpy (h_p, d_p, N*sizeof(particle), hipMemcpyDeviceToHost);
  cu_check(cuError, __FILE__, __LINE__);
  
  // save bins to file
  filename.insert(0, "../output/");
  filename.append(".dat");
  pFile = fopen(filename.c_str(), "w");
  
  for (int i = 0; i < N; i++) 
  {
    fprintf(pFile, " %d %d \n", i, int(h_p[i].y/ds));
  }
  
  fclose(pFile);

  //free host memory for particle vector
  free(h_p);
  
  return;
}

/******************** DEVICE KERNELS DEFINITIONS *********************/



/**********************************************************/
