/****************************************************************************
 *                                                                          *
 *    CUPIC is a code that simulates the interaction between plasma and     *
 *    a langmuir probe using PIC techniques accelerated with the use of     *
 *    GPU hardware (CUDA extension of C/C++)                                *
 *                                                                          *
 ****************************************************************************/


/****************************** HEADERS ******************************/

#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <gsl/gsl_rng.h>	  		//gsl library for random number generation
#include <gsl/gsl_randist.h>		//gsl library for random number generation

using namespace std;

#define PI 3.1415926535897932		//symbolic constant for PI

struct particle
{
  double x;
  double y;
  double vx;
  double vy;
}

/****************************** FUNCTION PROTOTIPES ******************************/

void initialize (double **h_qi, double **h_qe, double **h_mi, double **h_me, double **h_kti, double **h_kte, double **h_rho, double **h_phi, double **h_Ex, double **h_Ey, particle **h_e, particle **h_i, double **d_qi, double **d_qe, double **d_mi, double **d_me, double **d_kti, double **d_kte, double **d_rho, double **d_phi, double **d_Ex, double **d_Ey, particle **d_e, particle **d_i);

void read_particle_properties (double *h_qi, double *h_qe, double *h_mi, double *h_me, double *h_kti, double *h_kte);

/****************************** MAIN FUNCTION ******************************/

int main (int argc, const char* argv[])
{
  // host variables definition
  double *h_qi, *h_qe, *h_mi, *h_me, *h_kti, *h_kte;  //properties of particles
  double *h_rho, *h_phi, *h_Ex, *h_Ey;                //properties of mesh
  particle *h_e, *h_i;                                //vector of electron and ions

  // device variables definition
  double *d_qi, *d_qe, *d_mi, *d_me, *d_kti, *d_kte;  //properties of particles
  double *d_rho, *d_phi, *d_Ex, *d_Ey;                //properties of mesh
  particle *d_e, *d_i;                                //vector of electron and ions

  initilize (&h_qi, &h_qe, &h_mi, &h_me, &h_kti, &h_kte, &h_rho, &h_phi, &h_Ex, &h_Ey, &h_e, &h_i, &d_qi, &d_qe, &d_mi, &d_me, &d_kti, &d_kte, &d_rho, &d_phi, &d_Ex, &d_Ey, &d_e, &d_i);

  return 0;
}

/****************************** FUNCTION DEFINITION ******************************/

void initialize (double **h_qi, double **h_qe, double **h_mi, double **h_me, double **h_kti, double **h_kte, double **h_rho, double **h_phi, double **h_Ex, double **h_Ey, particle **h_e, particle **h_i, double **d_qi, double **d_qe, double **d_mi, double **d_me, double **d_kti, double **d_kte, double **d_rho, double **d_phi, double **d_Ex, double **d_Ey, particle **d_e, particle **d_i)
{
  // allocate host memory for particle properties
  *h_qi = (double*) malloc(sizeof(double));
  *h_qe = (double*) malloc(sizeof(double));
  *h_mi = (double*) malloc(sizeof(double));
  *h_me = (double*) malloc(sizeof(double));
  *h_kti = (double*) malloc(sizeof(double));
  *h_kte = (double*) malloc(sizeof(double));

  // read particle properties from input file
  read_particle_properties (*h_qi, *h_qe, *h_mi, *h_me, *h_kti, *h_kte);

  // allocate device memory for particle properties
  hipMalloc (*d_qi, sizeof(double));
  hipMalloc (*d_qe, sizeof(double));
  hipMalloc (*d_mi, sizeof(double));
  hipMalloc (*d_me, sizeof(double));
  hipMalloc (*d_kti, sizeof(double));
  hipMalloc (*d_kte, sizeof(double));

  // copy particle properties from host to device memory
  hipMemcpy (*d_qi, *h_qi, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_qe, *h_qe, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_mi, *h_mi, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_me, *h_me, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_kti, *h_kti, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_kte, *h_kte, sizeof(double), hipMemcpyHostToDevice);

  //show data
  cout << **h_qi << endl;
  cout << **h_qe << endl;
  cout << **h_mi << endl;
  cout << **h_me << endl;
  cout << **h_kti << endl;
  cout << **h_kte << endl;

  return; 
}

void read_particle_properties (double *h_qi, double *h_qe, double *h_mi, double *h_me, double *h_kti, double *h_kte)
{
  // function variables
  ifstream myfile;
  char line[80];

  // function body
  myfile.open("../input/input_data");
  if (myfile.is_open())
  {
    myfile.getline (line, 80);
    myfile.getline (line, 80);
    sscanf (line, "q_i = %lf \n", h_qi);
    myfile.getline (line, 80);
    sscanf (line, "q_e = %lf \n", h_qe);
    myfile.getline (line, 80);
    sscanf (line, "m_i = %lf \n", h_mi);
    myfile.getline (line, 80);
    sscanf (line, "m_e = %lf \n", h_me);
    myfile.getline (line, 80);
    sscanf (line, "kT_i = %lf \n", h_kti);
    myfile.getline (line, 80);
    sscanf (line, "kT_e = %lf \n", h_kte);
  } else
  {
    cout << "input data file could not be opened" << endl;
  }

  return; 
}
