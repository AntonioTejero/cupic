#include "hip/hip_runtime.h"
/****************************************************************************
 *                                                                          *
 *    CUPIC is a code that simulates the interaction between plasma and     *
 *    a langmuir probe using PIC techniques accelerated with the use of     *
 *    GPU hardware (CUDA extension of C/C++)                                *
 *                                                                          *
 ****************************************************************************/


/****************************** HEADERS ******************************/

#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <gsl/gsl_rng.h>        //gsl library for random number generation
#include <gsl/gsl_randist.h>    //gsl library for random number generation

using namespace std;

#define PI 3.1415926535897932		//symbolic constant for PI

struct particle
{
  double x;
  double y;
  double vx;
  double vy;
};

/****************************** FUNCTION PROTOTIPES ******************************/

void initialize (double **h_qi, double **h_qe, double **h_mi, double **h_me, double **h_kti, double **h_kte, double **h_phi_p, double **h_n, double **h_Lx, double **h_Ly, double **h_dx, double **h_dy, double **h_dz, double **h_t, double **h_dt, double **h_epsilon, double **h_rho, double **h_phi, double **h_Ex, double **h_Ey, particle **h_e, particle **h_i, double **d_qi, double **d_qe, double **d_mi, double **d_me, double **d_kti, double **d_kte, double **d_phi_p, double **d_n, double **d_Lx, double **d_Ly, double **d_dx, double **d_dy, double **d_dz, double **d_t, double **d_dt, double **d_epsilon, double **d_rho, double **d_phi, double **d_Ex, double **d_Ey, particle **d_e, particle **d_i);

void read_input_file (double *h_qi, double *h_qe, double *h_mi, double *h_me, double *h_kti, double *h_kte, double *h_phi_p, double *h_n, double *h_Lx, double *h_Ly, double *h_dx, double *h_dy, double *h_dz, double *h_t, double *h_dt, double *h_epsilon);

/****************************** MAIN FUNCTION ******************************/

int main (int argc, const char* argv[])
{
  // host variables definition
  double *h_qi, *h_qe, *h_mi, *h_me, *h_kti, *h_kte;  //properties of particles (charge, mass and temperature of particle species)
  double *h_n;                                        //plasma properties (plasma density)
  double *h_phi_p;                                    //probe properties (probe potential)
  double *h_Lx, *h_Ly, *h_dx, *h_dy, *h_dz;           //geometrical properties of simulation (simulation dimensions and spacial step)
  double *h_epsilon;                                  //electromagnetic properties
  double *h_rho, *h_phi, *h_Ex, *h_Ey;                //properties of mesh (charge density, potential and fields at point of the mesh)
  double *h_t, *h_dt;                                 //timing variables (simulation time and time step)
  particle *h_e, *h_i;                                //vector of electron and ions
  unsigned int *h_bookmarke;                          //vector that stores the endpoint of each particle bin (electrons)
  unsigned int *h_bookmarki;                          //vector that stores the endpoint of each particle bin (ions)

  // device variables definition
  double *d_qi, *d_qe, *d_mi, *d_me, *d_kti, *d_kte;  //properties of particles (charge, mass and temperature of particle species)
  double *d_n;                                        //plasma properties (plasma density)
  double *d_phi_p;                                    //probe properties (probe potential)
  double *d_Lx, *d_Ly, *d_dx, *d_dy, *d_dz;           //geometrical properties of simulation (simulation dimensions and spacial step)
  double *d_epsilon;                                  //electromagnetic properties
  double *d_rho, *d_phi, *d_Ex, *d_Ey;                //properties of mesh (charge density, potential and fields at point of the mesh)
  double *d_t, *d_dt;                                 //timing variables (simulation time and time step)
  particle *d_e, *d_i;                                //vector of electron and ions
  unsigned int *d_bookmarke;                          //vector that stores the endpoint of each particle bin (electrons)
  unsigned int *d_bookmarki;                          //vector that stores the endpoint of each particle bin (ions)

  initialize (&h_qi, &h_qe, &h_mi, &h_me, &h_kti, &h_kte, &h_phi_p, &h_n, &h_Lx, &h_Ly, &h_dx, &h_dy, &h_dz, &h_t, &h_dt, &h_epsilon, &h_rho, &h_phi, &h_Ex, &h_Ey, &h_e, &h_i, &h_bookmarke, &h_bookmarki, &d_qi, &d_qe, &d_mi, &d_me, &d_kti, &d_kte, &d_phi_p, &d_n, &d_Lx, &d_Ly, &d_dx, &d_dy, &d_dz, &d_t, &d_dt, &d_epsilon, &d_rho, &d_phi, &d_Ex, &d_Ey, &d_e, &d_i, &d_bookmarke, &d_bookmarki);

  return 0;
}

/****************************** FUNCTION DEFINITION ******************************/

void initialize (double **h_qi, double **h_qe, double **h_mi, double **h_me, double **h_kti, double **h_kte, double **h_phi_p, double **h_n, double **h_Lx, double **h_Ly, double **h_dx, double **h_dy, double **h_dz, double **h_t, double **h_dt, double **h_epsilon, double **h_rho, double **h_phi, double **h_Ex, double **h_Ey, particle **h_e, particle **h_i, double **d_qi, double **d_qe, double **d_mi, double **d_me, double **d_kti, double **d_kte, double **d_phi_p, double **d_n, double **d_Lx, double **d_Ly, double **d_dx, double **d_dy, double **d_dz, double **d_t, double **d_dt, double **d_epsilon, double **d_rho, double **d_phi, double **d_Ex, double **d_Ey, particle **d_e, particle **d_i)
{
  // function variables
  int N;                                          //initial number of particle of each species
  int ncx, ncy;                                   //number of grid points in each dimension
  gsl_rng * rng = gsl_rng_alloc(gsl_rng_default); //default random number generator (gsl)
  
  // initialize enviromental variables for gsl random number generator
  gsl_rng_env_setup();
  
  // allocate host memory for particle properties
  *h_qi = (double*) malloc(sizeof(double));
  *h_qe = (double*) malloc(sizeof(double));
  *h_mi = (double*) malloc(sizeof(double));
  *h_me = (double*) malloc(sizeof(double));
  *h_kti = (double*) malloc(sizeof(double));
  *h_kte = (double*) malloc(sizeof(double));
  
  // allocate host memory for plasma properties
  *h_n = (double*) malloc(sizeof(double));
  
  // allocate host memory for probe properties
  *h_phi_p = (double*) malloc(sizeof(double));
  
  // allocate host memory for geometrical properties of simulation
  *h_Lx = (double*) malloc(sizeof(double));
  *h_Ly = (double*) malloc(sizeof(double));
  *h_dx = (double*) malloc(sizeof(double));
  *h_dy = (double*) malloc(sizeof(double));
  *h_dz = (double*) malloc(sizeof(double));
  
  // allocate host memory for electromagnetic properties
  *h_epsilon = (double*) malloc(sizeof(double));
  
  // allocate host memory for mesh properties
  *h_rho = (double*) malloc(sizeof(double));
  *h_phi = (double*) malloc(sizeof(double));
  *h_Ex = (double*) malloc(sizeof(double));
  *h_Ey = (double*) malloc(sizeof(double));
  
  // allocate host memory for timing variables
  *h_t = (double*) malloc(sizeof(double));
  *h_dt = (double*) malloc(sizeof(double));
  
  // allocate device memory for particle properties
  hipMalloc (d_qi, sizeof(double));
  hipMalloc (d_qe, sizeof(double));
  hipMalloc (d_mi, sizeof(double));
  hipMalloc (d_me, sizeof(double));
  hipMalloc (d_kti, sizeof(double));
  hipMalloc (d_kte, sizeof(double));
  
  // allocate device memory for plasma properties
  hipMalloc (d_n, sizeof(double));
  
  // allocate device memory for probe properties
  hipMalloc (d_phi_p, sizeof(double));
  
  // allocate device memory for geometrical properties of simulation
  hipMalloc (d_Lx, sizeof(double));
  hipMalloc (d_Ly, sizeof(double));
  hipMalloc (d_dx, sizeof(double));
  hipMalloc (d_dy, sizeof(double));
  hipMalloc (d_dz, sizeof(double));
  
  // allocate device memory for electromagnetic properties
  hipMalloc (d_epsilon, sizeof(double));
  
  // allocate device memory for mesh properties
  hipMalloc (d_rho, sizeof(double));
  hipMalloc (d_phi, sizeof(double));
  hipMalloc (d_Ex, sizeof(double));
  hipMalloc (d_Ey, sizeof(double));
  
  // allocate device memory for timing variables
  hipMalloc (d_t, sizeof(double));
  hipMalloc (d_dt, sizeof(double));

  // read input file
  read_input_file (*h_qi, *h_qe, *h_mi, *h_me, *h_kti, *h_kte, *h_phi_p, *h_n, *h_Lx, *h_Ly, *h_dx, *h_dy, *h_dz, *h_t, *h_dt, *h_epsilon);
  
  // calculate initial number of particles and number of mesh points
  N = (**h_Lx)*(**h_dy)*(**h_dz)*(**h_n);
  ncx = (**h_Lx)/(**h_dx)+1;
  ncy = (**h_Ly)/(**h_dy)+1;
  N *= ncy;
  
  // allocate host memory for particle vectors
  *h_i = (particle*) malloc(N*sizeof(particle));
  *h_e = (particle*) malloc(N*sizeof(particle));
  
  // allocate host memory for bookmark vectors
  *h_bookmarke =  malloc((ncy-1)*sizeof(unsigned int));
  *h_bookmarki =  malloc((ncy-1)*sizeof(unsigned int));
  
  // allocate host memory for mesh variables
  *h_rho = (double*) malloc(ncx*ncy*sizeof(double));
  *h_phi = (double*) malloc(ncx*ncy*sizeof(double));
  *h_Ex = (double*) malloc(ncx*ncy*sizeof(double));
  *h_Ey = (double*) malloc(ncx*ncy*sizeof(double));
  
  // allocate device memory for particle vectors
  hipMalloc (d_i, N*sizeof(particle));
  hipMalloc (d_e, N*sizeof(particle));
  
  // allocate device memory for bookmark vectors
  hipMalloc (d_bookmarke, (ncy-1)*sizeof(unsigned int));
  hipMalloc (d_bookmarki, (ncy-1)*sizeof(unsigned int));
  
  // allocate device memory for mesh variables
  hipMalloc (d_rho, ncx*ncy*sizeof(double));
  hipMalloc (d_phi, ncx*ncy*sizeof(double));
  hipMalloc (d_Ex, ncx*ncy*sizeof(double));
  hipMalloc (d_Ey, ncx*ncy*sizeof(double));
  
  // initialize particle vectors and bookmarks (host memory)
  for (int i = 0; i < ncy-1; i++) 
  {
    (*h_bookmarke)[i] = (i+1)*N/(ncy-1);
    (*h_bookmarki)[i] = (i+1)*N/(ncy-1);
    for (int j = 0; j < N/(ncy-1); j++) 
    {
      // initialize ions
      (*h_i)[i*N/(ncy-1)+j].x = gsl_rng_uniform_pos(rng)*(**h_Lx);
      (*h_i)[i*N/(ncy-1)+j].y = double(i)*(**h_dy)+gsl_rng_uniform_pos(rng)*(**h_dy);
      (*h_i)[i*N/(ncy-1)+j].vx = gsl_ran_gaussian(rng, sqrt((**h_kti)/(**h_mi)));
      (*h_i)[i*N/(ncy-1)+j].vy = gsl_ran_gaussian(rng, sqrt((**h_kti)/(**h_mi)));
      
      // initialize electrons
      (*h_e)[i*N/(ncy-1)+j].x = gsl_rng_uniform_pos(rng)*(**h_Lx);
      (*h_e)[i*N/(ncy-1)+j].y = double(i)*(**h_dy)+gsl_rng_uniform_pos(rng)*(**h_dy);
      (*h_e)[i*N/(ncy-1)+j].vx = gsl_ran_gaussian(rng, sqrt((**h_kte)/(**h_me)));
      (*h_e)[i*N/(ncy-1)+j].vy = gsl_ran_gaussian(rng, sqrt((**h_kte)/(**h_me)));
    }
  }
  
  //initialize mesh variables (host memory)
  for (int im = 0; im < ncx; im++)
  {
    for (int jm = 0; jm < ncy; jm++)
    {
      (*h_Ex)[im+jm*(ncx)] = 0.0;
      (*h_Ey)[im+jm*(ncx)] = 0.0;
      (*h_rho)[im+jm*(ncx)] = 0.0;
      (*h_phi)[im+jm*(ncx)] = (1.0 - double(jm)/double(ncy-1))*(**h_phi_p);
    }
  }
  
  // copy particle properties from host to device memory
  hipMemcpy (*d_qi, *h_qi, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_qe, *h_qe, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_mi, *h_mi, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_me, *h_me, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_kti, *h_kti, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_kte, *h_kte, sizeof(double), hipMemcpyHostToDevice);

  // copy plasma properties from host to device memory
  hipMemcpy (*d_n, *h_n, sizeof(double), hipMemcpyHostToDevice);

  // copy probe properties from host to device memory
  hipMemcpy (*d_phi_p, *h_phi_p, sizeof(double), hipMemcpyHostToDevice);
  
  // copy geometrical properties from host to device memory
  hipMemcpy (*d_Lx, *h_Lx, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_Ly, *h_Ly, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_dx, *h_dx, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_dy, *h_dy, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_dz, *h_dz, sizeof(double), hipMemcpyHostToDevice);

  // copy electromagnetic properties from host to device memory
  hipMemcpy (*d_epsilon, *h_epsilon, sizeof(double), hipMemcpyHostToDevice);
  
  // copy mesh properties from host to device memory
  hipMemcpy (*d_rho, *h_rho, ncx*ncy*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_phi, *h_phi, ncx*ncy*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_Ex, *h_Ex, ncx*ncy*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_Ey, *h_Ey, ncx*ncy*sizeof(double), hipMemcpyHostToDevice);
  
  // copy timing variables from host to device memory
  hipMemcpy (*d_t, *h_t, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_dt, *h_dt, sizeof(double), hipMemcpyHostToDevice);
  
  // copy particle and bookmark vectors from host to device memory
  hipMemcpy (*d_i, *h_i, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_e, *h_e, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy (*d_bookmarki, *h_bookmarki, (ncy-1)*sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy (*d_bookmarke, *h_bookmarke, (ncy-1)*sizeof(unsigned int), hipMemcpyHostToDevice);
  
  return;
}

/**********************************************************/

void read_input_file (double *h_qi, double *h_qe, double *h_mi, double *h_me, double *h_kti, double *h_kte, double *h_phi_p, double *h_n, double *h_Lx, double *h_Ly, double *h_dx, double *h_dy, double *h_dz, double *h_t, double *h_dt, double *h_epsilon)
{
  // function variables
  ifstream myfile;
  char line[80];

  // function body
  myfile.open("../input/input_data");
  if (myfile.is_open())
  {
    myfile.getline (line, 80);
    myfile.getline (line, 80);
    sscanf (line, "q_i = %lf \n", h_qi);
    myfile.getline (line, 80);
    sscanf (line, "q_e = %lf \n", h_qe);
    myfile.getline (line, 80);
    sscanf (line, "m_i = %lf \n", h_mi);
    myfile.getline (line, 80);
    sscanf (line, "m_e = %lf \n", h_me);
    myfile.getline (line, 80);
    sscanf (line, "kT_i = %lf \n", h_kti);
    myfile.getline (line, 80);
    sscanf (line, "kT_e = %lf \n", h_kte);
    myfile.getline (line, 80);
    myfile.getline (line, 80);
    myfile.getline (line, 80);
    sscanf (line, "phi_p = %lf \n", h_phi_p);
    myfile.getline (line, 80);
    myfile.getline (line, 80);
    myfile.getline (line, 80);
    sscanf (line, "n = %lf \n", h_n);
    myfile.getline (line, 80);
    myfile.getline (line, 80);
    myfile.getline (line, 80);
    sscanf (line, "Lx = %lf \n", h_Lx);
    myfile.getline (line, 80);
    sscanf (line, "Ly = %lf \n", h_Ly);
    myfile.getline (line, 80);
    sscanf (line, "dx = %lf \n", h_dx);
    myfile.getline (line, 80);
    sscanf (line, "dy = %lf \n", h_dy);
    myfile.getline (line, 80);
    sscanf (line, "dz = %lf \n", h_dz);
    myfile.getline (line, 80);
    myfile.getline (line, 80);
    myfile.getline (line, 80);
    sscanf (line, "t = %lf \n", h_t);
    myfile.getline (line, 80);
    sscanf (line, "dt = %lf \n", h_dt);
    myfile.getline (line, 80);
    myfile.getline (line, 80);
    myfile.getline (line, 80);
    sscanf (line, "epsilon0 = %lf \n", h_epsilon);
  } else
  {
    cout << "input data file could not be opened" << endl;
    exit(1);
  }

  return;
}

/**********************************************************/

// fast particle-to-grid interpolation (based on the article by George Stantchev, William Dorland and Nail Gumerov)

void fast_particle_to_grid_interpolation ()
{
  // function variables
  
  // function body
//   particle_bining();
//   particle_to_cell_density_deposition();
//   cell_to_vertex_density_accumulation();
  
  return;
}

/**********************************************************/

void particle_bining()
{
  // function variables
  
  // function body
//   particle_defragmentation();
//   particle_rebracketing();
  
  return;
}

/**********************************************************/

void particle_defragmentation(int bin_start, int bin_end, double dy, int bin, particle * p) 
{
  // kernel shared memory
  __shared__ particle p_sha[blockDim.x];
  __shared__ int tail = 0;
  // kernel registers
  particle p_reg, p_dummy;
  int i = bin_start;
  int i_shifted = bin_start + blockDim.x;
  int new_bin;
  int swap_index;
  
  /*--------------------------- function body ---------------------------*/
  
  //---- cleaning first batch of particles
  
  // reading from global memory
  p_sha[threadIdx.x] = p[i_shifted+threadIdx.x];
  __syncthreads();
  p_reg = p[i+threadIdx.x];
  
  // obtaining valid swap_index for each "-" particle in first batch
  new_bin = p_reg.y/dy;
  if (new_bin<bin)
  {
    do
    {
      swap_index = atomicAdd(&tail, 1);
    } while (int(p_sha[swap_index].y/dy)<bin);
  }
  __syncthreads();
  
  // swapping "-" particles from first batch with "non -" particles from second batch
  if (new_bin<bin)
  {
    p_dummy = p_reg;
    p_reg = p_sha[swap_index];
    p_sha[swap_index] = dummy;
  }
  __syncthreads();
  
  // write back particle batches to global memory
  p[i+threadIdx.x] = p_reg;                       
  __syncthreads();
  p[i_shifted+threadIdx.x] = p_sha[threadIdx.x];
  __syncthreads();
  
  // reset tail parameter (shared memory)
  if (threadIdx.x ==1)
  {
    tail = 0;
  }
  
  //---- start of "-" defrag algorithm
  
  while (i_shifted+blockDim.x<=bin_end)
  {
    // read exchange queue from global memory
    p_sha[threadIdx.x] = p[i+threadIdx.x];
    __syncthreads();
    
    // read batch of particles to be analyzed from global memory
    p_reg = p[i_shifted+threadIdx.x];
    
    // analyze batch of particle in registers
    new_bin = p_reg.y/dy;
    if (new_bin<bin)
    {
      swap_index = atomicAdd(&tail, 1);
    }
    __syncthreads()
    
    // swapping "-" particles from registers with particles in exchange queue (shared memory)
    if (new_bin<bin)
    {
      p_dummy = p_reg;
      p_reg = p_sha[swap_index];
      p_sha[swap_index] = dummy;
    }
    __syncthreads();
    
    // write back particle batches to global memory
    p[i_shifted+threadIdx.x] = p_reg;                       
    __syncthreads();
    p[i+threadIdx.x] = p_sha[threadIdx.x];
    __syncthreads();
    
    // update batches parameters for next iteration
    i += tail;
    i_shifted += blockDim.x;
    // reset tail parameter (shared memory)
    if (threadIdx.x ==1)
    {
      tail = 0;
    }
    
  }
  
  //---- defrag of last stride (incomplete) of the bin
  
  if (i_shifted+threadIdx.x<=bin_end)
  {
    // read exchange queue from global memory
    p_sha[threadIdx.x] = p[i+threadIdx.x];
  }
  __syncthreads();
  
  if (i_shifted+threadIdx.x<=bin_end)
  {
    // read batch of particles to be analyzed from global memory
    p_reg = p[i_shifted+threadIdx.x];
    
    // analyze batch of particle in registers
    new_bin = p_reg.y/dy;
    if (new_bin<bin)
    {
      swap_index = atomicAdd(&tail, 1);
    }
  }
  __syncthreads();
  
  if (i_shifted+threadIdx.x<=bin_end)
  {
    // swapping "-" particles from registers with particles in exchange queue (shared memory)
    if (new_bin<bin)
    {
      p_dummy = p_reg;
      p_reg = p_sha[swap_index];
      p_sha[swap_index] = dummy;
    }
  }
  __syncthreads();
  
  // write back particle batches to global memory
  if (i_shifted+threadIdx.x<=bin_end)
  {
    p[i_shifted+threadIdx.x] = p_reg;                       
  }
  __syncthreads();
  if (i_shifted+threadIdx.x<=bin_end)
  {
    p[i+threadIdx.x] = p_sha[threadIdx.x];
  }
  __syncthreads();
  
  return;
}

/**********************************************************/

void particle_rebracketing() 
{
  // function varibales
  
  // functiona body
  
  
  return;
}




/**********************************************************/
